#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void invertColors(unsigned char *image, int width, int height, int channels)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Make sure we don't access memory outside the image.
    if (x < width && y < height)
    {
        int idx = (y * width + x) * channels;  // Compute the index for the pixel.


        for (int c = 0; c < channels; ++c)
        {
            image[idx + c] = 255 - image[idx + c];
        }
    }
}

int main(int argc, char *argv[])
{

    int width = 512;
    int height = 512;
    int channels = 3; // RGB image.
    size_t imageSize = width * height * channels * sizeof(unsigned char);
    
    // Allocate host memory for the image.
    unsigned char *h_image = (unsigned char*)malloc(imageSize);
    if (h_image == NULL) {
        fprintf(stderr, "Error: Failed to allocate host memory.\n");
        return EXIT_FAILURE;
    }

    // Initialize the dummy image with a constant value (e.g., 100 for each channel).
    for (int i = 0; i < width * height * channels; i++) {
        h_image[i] = 100;
    }
    
    // Allocate device memory.
    unsigned char *d_image;
    hipError_t err = hipMalloc((void**)&d_image, imageSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: Failed to allocate device memory: %s\n", hipGetErrorString(err));
        free(h_image);
        return EXIT_FAILURE;
    }
    
    // Copy the image data from the host to the device.
    err = hipMemcpy(d_image, h_image, imageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: Failed to copy image from host to device: %s\n", hipGetErrorString(err));
        hipFree(d_image);
        free(h_image);
        return EXIT_FAILURE;
    }
    
    // Define the block and grid dimensions.
    // Using 16x16 threads per block is a common choice.
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);
    
    // Launch the kernel.
    invertColors<<<gridSize, blockSize>>>(d_image, width, height, channels);
    
    // Check if the kernel launch resulted in an error.
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error: Failed to launch kernel: %s\n", hipGetErrorString(err));
        hipFree(d_image);
        free(h_image);
        return EXIT_FAILURE;
    }
    
    // Copy the processed image data back from the device to the host.
    err = hipMemcpy(h_image, d_image, imageSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: Failed to copy result from device to host: %s\n", hipGetErrorString(err));
        hipFree(d_image);
        free(h_image);
        return EXIT_FAILURE;
    }
    

    printf("First 10 pixels (R, G, B):\n");
    for (int i = 0; i < 10; i++) {
        int index = i * channels;
        printf("Pixel %d: %3u, %3u, %3u\n", i, h_image[index], h_image[index+1], h_image[index+2]);
    }
    
    // Clean up device and host memory.
    hipFree(d_image);
    free(h_image);
    
    printf("Processing complete.\n");
    return EXIT_SUCCESS;
}
